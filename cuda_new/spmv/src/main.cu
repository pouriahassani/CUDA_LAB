#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCK_SIZE 256

typedef struct
{
    int *row_indices;
    int *col_indices;
    float *values;
    int num_nonzeros;
    int num_rows;
    int num_cols;
} COOMatrix;

__global__ void SpMVKernelCOO(int *d_row_indices, int *d_col_indices, float *d_values, float *d_x, float *d_y, int num_nonzeros)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < num_nonzeros)
    {
        atomicAdd(&d_y[d_row_indices[i]], d_values[i] * d_x[d_col_indices[i]]);
    }
}

__global__ void SpMVKernelNaive(float *d_A, float *d_x, float *d_y, int num_rows, int num_cols)
{
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    if (row < num_rows)
    {
        float dot_product = 0;
        for (int col = 0; col < num_cols; col++)
        {
            dot_product += d_A[row * num_cols + col] * d_x[col];
        }
        d_y[row] = dot_product;
    }
}

void generateRandomCOOMatrix(COOMatrix *cooMatrix, int num_rows, int num_cols, int num_nonzeros)
{
    cooMatrix->num_rows = num_rows;
    cooMatrix->num_cols = num_cols;
    cooMatrix->num_nonzeros = num_nonzeros;
    cooMatrix->row_indices = (int *)malloc(num_nonzeros * sizeof(int));
    cooMatrix->col_indices = (int *)malloc(num_nonzeros * sizeof(int));
    cooMatrix->values = (float *)malloc(num_nonzeros * sizeof(float));

    for (int i = 0; i < num_nonzeros; i++)
    {
        cooMatrix->row_indices[i] = rand() % num_rows;
        cooMatrix->col_indices[i] = rand() % num_cols;
        cooMatrix->values[i] = (float)(rand() % 100) / 10.0;
    }
}

void generateRandomDenseMatrix(float *A, int num_rows, int num_cols)
{
    for (int i = 0; i < num_rows * num_cols; i++)
    {
        A[i] = (float)(rand() % 100) / 10.0;
    }
}

void freeCOOMatrix(COOMatrix *cooMatrix)
{
    free(cooMatrix->row_indices);
    free(cooMatrix->col_indices);
    free(cooMatrix->values);
}

void SpMVCOO(int *d_row_indices, int *d_col_indices, float *d_values, float *d_x, float *d_y, int num_nonzeros, hipStream_t stream, float *kernel_time)
{
    int numBlocks = (num_nonzeros + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);

    SpMVKernelCOO<<<numBlocks, BLOCK_SIZE, 0, stream>>>(d_row_indices, d_col_indices, d_values, d_x, d_y, num_nonzeros);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);

    hipEventElapsedTime(kernel_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

void SpMVNaive(float *d_A, float *d_x, float *d_y, int num_rows, int num_cols, hipStream_t stream, float *kernel_time)
{
    int numBlocks = (num_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, stream);

    SpMVKernelNaive<<<numBlocks, BLOCK_SIZE, 0, stream>>>(d_A, d_x, d_y, num_rows, num_cols);

    hipEventRecord(stop, stream);
    hipEventSynchronize(stop);

    hipEventElapsedTime(kernel_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

double get_time_in_ms()
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec * 1000.0 + ts.tv_nsec / 1000000.0;
}

int main(int argc, char *argv[])
{
    if (argc != 4)
    {
        fprintf(stderr, "Usage: %s <num_rows> <num_cols> <num_nonzeros>\n", argv[0]);
        return -1;
    }

    int num_rows = atoi(argv[1]);
    int num_cols = atoi(argv[2]);
    int num_nonzeros = atoi(argv[3]);

    srand(time(NULL));

    // Generate random matrices
    COOMatrix cooMatrix;
    generateRandomCOOMatrix(&cooMatrix, num_rows, num_cols, num_nonzeros);

    float *A = (float *)malloc(num_rows * num_cols * sizeof(float));
    generateRandomDenseMatrix(A, num_rows, num_cols);

    float *x = (float *)malloc(num_cols * sizeof(float));
    float *y = (float *)malloc(num_rows * sizeof(float));
    float *y_naive = (float *)malloc(num_rows * sizeof(float));
    for (int i = 0; i < num_cols; i++)
    {
        x[i] = (float)(rand() % 100) / 10.0;
    }

    int *d_row_indices, *d_col_indices;
    float *d_values, *d_A, *d_x, *d_y;

    hipMalloc((void **)&d_row_indices, cooMatrix.num_nonzeros * sizeof(int));
    hipMalloc((void **)&d_col_indices, cooMatrix.num_nonzeros * sizeof(int));
    hipMalloc((void **)&d_values, cooMatrix.num_nonzeros * sizeof(float));
    hipMalloc((void **)&d_A, num_rows * num_cols * sizeof(float));
    hipMalloc((void **)&d_x, num_cols * sizeof(float));
    hipMalloc((void **)&d_y, num_rows * sizeof(float));

    hipMemcpy(d_row_indices, cooMatrix.row_indices, cooMatrix.num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, cooMatrix.col_indices, cooMatrix.num_nonzeros * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, cooMatrix.values, cooMatrix.num_nonzeros * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_A, A, num_rows * num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, num_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, num_rows * sizeof(float));

    hipStream_t stream;
    hipStreamCreate(&stream);

    float kernel_time_coo = 0;
    float kernel_time_naive = 0;
    double start_time, end_time, execution_time_coo, execution_time_naive;

    // COO version
    start_time = get_time_in_ms();
    SpMVCOO(d_row_indices, d_col_indices, d_values, d_x, d_y, cooMatrix.num_nonzeros, stream, &kernel_time_coo);
    hipStreamSynchronize(stream);
    end_time = get_time_in_ms();
    execution_time_coo = end_time - start_time;

    hipMemcpyAsync(y, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    // Naive version
    hipMemset(d_y, 0, num_rows * sizeof(float)); // Reset the output vector for the naive version

    start_time = get_time_in_ms();
    SpMVNaive(d_A, d_x, d_y, num_rows, num_cols, stream, &kernel_time_naive);
    hipStreamSynchronize(stream);
    end_time = get_time_in_ms();
    execution_time_naive = end_time - start_time;

    hipMemcpyAsync(y_naive, d_y, num_rows * sizeof(float), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    // Validate results
    for (int i = 0; i < 10; i++)
    {
        printf("y_coo[%d] = %f, y_naive[%d] = %f\n", i, y[i], i, y_naive[i]);
    }

    FILE *f = fopen("res.csv", "a");
    if (f == NULL)
    {
        fprintf(stderr, "Error opening file for writing\n");
        return -1;
    }
    fprintf(f, "%d,%d,%d,%f,%f,%f,%f\n", num_rows, num_cols, num_nonzeros, execution_time_coo, kernel_time_coo, execution_time_naive, kernel_time_naive);
    fclose(f);

    free(x);
    free(y);
    free(y_naive);
    free(A);
    freeCOOMatrix(&cooMatrix);

    hipFree(d_row_indices);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);

    hipStreamDestroy(stream);

    return 0;
}
